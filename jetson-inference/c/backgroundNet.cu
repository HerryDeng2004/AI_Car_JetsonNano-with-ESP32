#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include "cudaFilterMode.cuh"
#include "cudaVector.h"

#include "imageFormat.h"


//#define RETAIN_ALPHA


// gpuBackgroundMask
template<typename T, cudaFilterMode filter, bool mask_alpha>
__global__ void gpuBackgroundMask( T* input, T* output, int width, int height, float* mask, int mask_width, int mask_height )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = y * width + x;
	
	if( x >= width || y >= height )
		return;

	float mask_px = cudaFilterPixel<filter>(mask, x, y, mask_width, mask_height, width, height);
	
	if( mask_px < 0 ) mask_px = 0;
	if( mask_px > 1 ) mask_px = 1;
	
	if( !mask_alpha )
	{
		// retain the original alpha channel
		const float4 input_px = cast_vec<float4>(input[i]);
		
		const float4 output_px = make_float4(input_px.x * mask_px,
									  input_px.y * mask_px,
									  input_px.z * mask_px,
									  input_px.w);
		
		output[i] = cast_vec<T>(output_px);
	}
	else
	{
		// apply the mask to the alpha channel too
		output[i] = input[i] * mask_px;
	}
}


// cudaBackgroundMask
hipError_t cudaBackgroundMask( void* input, void* output, uint32_t width, uint32_t height, imageFormat format,
						  float* mask, uint32_t mask_width, uint32_t mask_height, bool mask_alpha,
						  cudaFilterMode filter, hipStream_t stream )
{
	if( !input || !output || !mask )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 || mask_width == 0 || mask_height == 0 )
		return hipErrorInvalidValue;
		
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	#define backgroundMaskFilter(type, maskAlpha, filterMode) \
		gpuBackgroundMask<type, maskAlpha, filterMode><<<gridDim, blockDim, 0, stream>>>( \
							(type*)input, (type*)output, width, height, \
							mask, mask_width, mask_height);

	#define backgroundMaskKernel(type) \
	{ \
		if( filter == FILTER_POINT && mask_alpha ) \
			backgroundMaskFilter(type, FILTER_POINT, true) \
		else if( filter == FILTER_POINT && !mask_alpha ) \
			backgroundMaskFilter(type, FILTER_POINT, false) \
		else if( filter == FILTER_LINEAR && mask_alpha ) \
			backgroundMaskFilter(type, FILTER_LINEAR, true) \
		else if( filter == FILTER_LINEAR && !mask_alpha ) \
			backgroundMaskFilter(type, FILTER_LINEAR, false) \
	}

	if( format == IMAGE_RGB8 )
		backgroundMaskKernel(uchar3)
	else if( format == IMAGE_RGBA8 )
		backgroundMaskKernel(uchar4)
	else if( format == IMAGE_RGB32F )
		backgroundMaskKernel(float3)
	else if( format == IMAGE_RGBA32F )
		backgroundMaskKernel(float4)
	else
	{
		imageFormatErrorMsg(LOG_CUDA, "cudaBackgroundMask()", format);
		return hipErrorInvalidValue;
	}
		
	return CUDA(hipGetLastError());
}


#if 0
	// note:  this seems unnecessary, because the mask min/max are almost always 0.0-1.0 anyways
	float mask_min = 1000000.0f;
	float mask_max = -1000000.0f;
	
	const uint32_t mask_width = GetOutputWidth();
	const uint32_t mask_height = GetOutputHeight();
	
	float* mask_ptr = mOutputs[0].CUDA;
	
	for( uint32_t y=0; y < mask_height; y++ )
	{
		for( uint32_t x=0; x < mask_width; x++ )
		{
			const float px = mask_ptr[y * mask_width + x];
			
			if( px < mask_min )
				mask_min = px;
			else if( px > mask_max )
				mask_max = px;
		}
	}
	
	printf("mask min=%f  max=%f\n", mask_min, mask_max);
	
	for( uint32_t y=0; y < mask_height; y++ )
		for( uint32_t x=0; x < mask_width; x++ )
			mask_ptr[y * mask_width + x] = (mask_ptr[y * mask_width + x] - mask_min) / (mask_max - mask_min);
#endif
